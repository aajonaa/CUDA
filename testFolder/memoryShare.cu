
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define VECTOR_SIZE 100000000 // Change this value to modify the vector size

// CUDA kernel to calculate dot product using shared memory
__global__ void dotProductCUDA(double* a, double* b, double* result) {
    __shared__ double temp[256]; // Shared memory for each block
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int localIndex = threadIdx.x;

    double tempResult = 0.0;
    while (tid < VECTOR_SIZE) {
        tempResult += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    temp[localIndex] = tempResult;

    // Synchronize within the block
    __syncthreads();

    // Reduction in shared memory
    int i = blockDim.x / 2;
    while (i != 0) {
        if (localIndex < i) {
            temp[localIndex] += temp[localIndex + i];
        }
        __syncthreads();
        i /= 2;
    }

    // Store result to global memory
    if (localIndex == 0) {
        result[blockIdx.x] = temp[0];
    }
}

int main() {
    srand(time(NULL)); // Seed for random number generation

    double *vectorA, *vectorB, *result;
    vectorA = (double*)malloc(VECTOR_SIZE * sizeof(double));
    vectorB = (double*)malloc(VECTOR_SIZE * sizeof(double));
    result = (double*)malloc(256 * sizeof(double)); // 256 threads per block

    if (vectorA == NULL || vectorB == NULL || result == NULL) {
        printf("Memory allocation failed.\n");
        return -1;
    }

    // Generating random values for the vectors
    for (int i = 0; i < VECTOR_SIZE; ++i) {
        vectorA[i] = (double)rand() / RAND_MAX; // Generating random values between 0 and 1
        vectorB[i] = (double)rand() / RAND_MAX;
    }

    double *dev_a, *dev_b, *dev_result;

    // Allocate memory on the device
    hipMalloc((void**)&dev_a, VECTOR_SIZE * sizeof(double));
    hipMalloc((void**)&dev_b, VECTOR_SIZE * sizeof(double));
    hipMalloc((void**)&dev_result, 256 * sizeof(double));

    // Copy input arrays from host to device
    hipMemcpy(dev_a, vectorA, VECTOR_SIZE * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, vectorB, VECTOR_SIZE * sizeof(double), hipMemcpyHostToDevice);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch kernel
    dotProductCUDA<<<256, 256>>>(dev_a, dev_b, dev_result);

    // Copy result array from device to host
    hipMemcpy(result, dev_result, 256 * sizeof(double), hipMemcpyDeviceToHost);

    // Final reduction on CPU
    double finalResult = 0.0;
    for (int i = 0; i < 256; ++i) {
        finalResult += result[i];
    }

    // Measure and print the execution time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Dot product: %f\n", finalResult);
    printf("Execution time: %f milliseconds\n", milliseconds);

    // Free allocated memory
    free(vectorA);
    free(vectorB);
    free(result);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_result);

    return 0;
}
