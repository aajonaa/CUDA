#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define VECTOR_SIZE 100000000 // Modify this value as needed

// Atomic add for double precision values
__device__ double atomicAddDouble(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);

    return __longlong_as_double(old);
}

// CPU version for dot product calculation
double dotProductCPU(double* a, double* b, int size) {
    double result = 0.0;
    for (int i = 0; i < size; ++i) {
        result += a[i] * b[i];
    }
    return result;
}

// CUDA kernel to calculate dot product using global memory for double arrays
__device__ double atomicAddDouble(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);

    return __longlong_as_double(old);
}

__global__ void dotProductGlobalMemory(double* a, double* b, double* result, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    double tempResult = 0.0;

    while (tid < size) {
        tempResult += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    atomicAddDouble(result, tempResult);
}

// CUDA kernel to calculate dot product using shared memory for double arrays
__global__ void dotProductSharedMemory(double* a, double* b, double* result, int size) {
    __shared__ double temp[256]; // Shared memory for each block
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int localIndex = threadIdx.x;

    double tempResult = 0.0;
    while (tid < size) {
        tempResult += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    temp[localIndex] = tempResult;

    // Synchronize within the block
    __syncthreads();

    // Reduction in shared memory
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (localIndex < i) {
            temp[localIndex] += temp[localIndex + i];
        }
        __syncthreads();
    }

    // Store result to global memory with atomic operation
    if (localIndex == 0) {
        atomicAddDouble(result, temp[0]);
    }
}

int main() {
    srand(time(NULL)); // Seed for random number generation

    double *vectorA, *vectorB, *resultCPU, *resultGlobalMemory, *resultSharedMemory;
    vectorA = (double*)malloc(VECTOR_SIZE * sizeof(double));
    vectorB = (double*)malloc(VECTOR_SIZE * sizeof(double));
    resultCPU = (double*)malloc(sizeof(double)); // For CPU version
    resultGlobalMemory = (double*)malloc(sizeof(double)); // For global memory version
    resultSharedMemory = (double*)malloc(sizeof(double)); // For shared memory version

    if (vectorA == NULL || vectorB == NULL || resultCPU == NULL || resultGlobalMemory == NULL || resultSharedMemory == NULL) {
        printf("Memory allocation failed.\n");
        return -1;
    }

    // Generating random values for the vectors
    for (int i = 0; i < VECTOR_SIZE; ++i) {
        vectorA[i] = (double)rand() / RAND_MAX; // Generating random values between 0 and 1
        vectorB[i] = (double)rand() / RAND_MAX;
    }

    // CPU version timing
    clock_t cpu_start = clock();
    *resultCPU = dotProductCPU(vectorA, vectorB, VECTOR_SIZE);
    clock_t cpu_end = clock();
    double cpu_time = ((double)(cpu_end - cpu_start)) / CLOCKS_PER_SEC * 1000.0; // in milliseconds

    // CUDA initialization and memory allocation for global memory version
    double *dev_a, *dev_b, *dev_resultGlobalMemory, *dev_resultSharedMemory;
    hipMalloc((void**)&dev_a, VECTOR_SIZE * sizeof(double));
    hipMalloc((void**)&dev_b, VECTOR_SIZE * sizeof(double));
    hipMalloc((void**)&dev_resultGlobalMemory, sizeof(double));
    hipMalloc((void**)&dev_resultSharedMemory, sizeof(double)); // Allocate memory for shared memory version
    hipMemcpy(dev_a, vectorA, VECTOR_SIZE * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, vectorB, VECTOR_SIZE * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(dev_resultGlobalMemory, 0, sizeof(double));
    hipMemset(dev_resultSharedMemory, 0, sizeof(double)); // Initialize result memory for shared memory version

    int blockSize = 256;
    int numBlocks = (VECTOR_SIZE + blockSize - 1) / blockSize;

    // Timing for global memory version
    clock_t global_start = clock();
    dotProductGlobalMemory<<<numBlocks, blockSize>>>(dev_a, dev_b, dev_resultGlobalMemory, VECTOR_SIZE);
    hipDeviceSynchronize();
    hipMemcpy(resultGlobalMemory, dev_resultGlobalMemory, sizeof(double), hipMemcpyDeviceToHost);
    clock_t global_end = clock();
    double global_time = ((double)(global_end - global_start)) / CLOCKS_PER_SEC * 1000.0; // in milliseconds

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_resultGlobalMemory);

    // CUDA initialization and memory allocation for shared memory version
    hipMalloc((void**)&dev_a, VECTOR_SIZE * sizeof(double));
    hipMalloc((void**)&dev_b, VECTOR_SIZE * sizeof(double));
    hipMemcpy(dev_a, vectorA, VECTOR_SIZE * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, vectorB, VECTOR_SIZE * sizeof(double), hipMemcpyHostToDevice);

    // Timing for shared memory version
    clock_t shared_start = clock();
    dotProductSharedMemory<<<numBlocks, blockSize>>>(dev_a, dev_b, dev_resultSharedMemory, VECTOR_SIZE);
    hipDeviceSynchronize();
    hipMemcpy(resultSharedMemory, dev_resultSharedMemory, sizeof(double), hipMemcpyDeviceToHost);
    clock_t shared_end = clock();
    double shared_time = ((double)(shared_end - shared_start)) / CLOCKS_PER_SEC * 1000.0; // in milliseconds

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_resultSharedMemory);

    // Printing results and timing information
    printf("CPU Dot Product Result: %f\n", *resultCPU);
    printf("Global Memory Dot Product Result: %f\n", *resultGlobalMemory);
    printf("Shared Memory Dot Product Result: %f\n", *resultSharedMemory);
    printf("CPU Time: %f ms\n", cpu_time);
    printf("Global Memory Time: %f ms\n", global_time);
    printf("Shared Memory Time: %f ms\n", shared_time);

    // Free allocated memory
    free(vectorA);
    free(vectorB);
    free(resultCPU);
    free(resultGlobalMemory);
    free(resultSharedMemory);

    return 0;
}
