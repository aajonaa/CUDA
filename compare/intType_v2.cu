
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define VECTOR_SIZE 100000000 // 设置向量大小

// CPU 计算
int dotProductCPU(int* a, int* b) {
    int result = 0;
    for (int i = 0; i < VECTOR_SIZE; ++i) {
        result += a[i] * b[i];
    }
    return result;
}

// 全局共享内存计算
__global__ void dotProductGlobalMemory(int* a, int* b, int* result) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int tempResult = 0;

    while (tid < VECTOR_SIZE) {
        tempResult += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    result[blockIdx.x * blockDim.x + threadIdx.x] = tempResult;
}

// 共享内存计算
__global__ void dotProductSharedMemory(int* a, int* b, int* result) {
    __shared__ int temp[256]; // Shared memory for each block
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int localIndex = threadIdx.x;

    int tempResult = 0;
    while (tid < VECTOR_SIZE) {
        tempResult += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    temp[localIndex] = tempResult;

    // 同步块
    __syncthreads();

    // 减少共享内存
    int i = blockDim.x / 2;
    while (i != 0) {
        if (localIndex < i) {
            temp[localIndex] += temp[localIndex + i];
        }
        __syncthreads();
        i /= 2;
    }

    // 存结果到全局内存
    if (localIndex == 0) {
        result[blockIdx.x] = temp[0];
    }
}

int main() {
    srand(time(NULL)); // 随机数生成

    int *vectorA, *vectorB, *result;
    vectorA = (int*)malloc(VECTOR_SIZE * sizeof(int));
    vectorB = (int*)malloc(VECTOR_SIZE * sizeof(int));
    result = (int*)malloc(256 * sizeof(int)); // 每块256个线程

    if (vectorA == NULL || vectorB == NULL || result == NULL) {
        printf("Memory allocation failed.\n");
        return -1;
    }

    // 生成向量
    for (int i = 0; i < VECTOR_SIZE; ++i) {
        vectorA[i] = rand() % 100; // 每维控制在100内
        vectorB[i] = rand() % 100;
    }

    // CPU version timing
    clock_t cpu_start = clock();
    int cpuResult = dotProductCPU(vectorA, vectorB);
    clock_t cpu_end = clock();
    double cpu_time = ((double)(cpu_end - cpu_start)) / CLOCKS_PER_SEC * 1000.0; // in milliseconds
    printf("CPU Dot Product Result: %d\n", cpuResult);
    printf("CPU Time: %f ms\n", cpu_time);

    // Global memory version timing
    int *dev_a, *dev_b, *dev_result;
    hipMalloc((void**)&dev_a, VECTOR_SIZE * sizeof(int));
    hipMalloc((void**)&dev_b, VECTOR_SIZE * sizeof(int));
    hipMalloc((void**)&dev_result, 256 * sizeof(int));
    hipMemcpy(dev_a, vectorA, VECTOR_SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, vectorB, VECTOR_SIZE * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t global_start, global_stop;
    hipEventCreate(&global_start);
    hipEventCreate(&global_stop);
    hipEventRecord(global_start);

    dotProductGlobalMemory<<<256, 256>>>(dev_a, dev_b, dev_result);

    hipMemcpy(result, dev_result, 256 * sizeof(int), hipMemcpyDeviceToHost);

    hipEventRecord(global_stop);
    hipEventSynchronize(global_stop);
    float global_milliseconds = 0;
    hipEventElapsedTime(&global_milliseconds, global_start, global_stop);

    printf("Global Memory Dot Product Result: %d\n", result[0]);
    printf("Global Memory Time: %f milliseconds\n", global_milliseconds);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_result);

    // Shared memory version timing
    hipMalloc((void**)&dev_a, VECTOR_SIZE * sizeof(int));
    hipMalloc((void**)&dev_b, VECTOR_SIZE * sizeof(int));
    hipMalloc((void**)&dev_result, 256 * sizeof(int));
    hipMemcpy(dev_a, vectorA, VECTOR_SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, vectorB, VECTOR_SIZE * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t shared_start, shared_stop;
    hipEventCreate(&shared_start);
    hipEventCreate(&shared_stop);
    hipEventRecord(shared_start);

    dotProductSharedMemory<<<256, 256>>>(dev_a, dev_b, dev_result);

    hipMemcpy(result, dev_result, 256 * sizeof(int), hipMemcpyDeviceToHost);

    hipEventRecord(shared_stop);
    hipEventSynchronize(shared_stop);
    float shared_milliseconds = 0;
    hipEventElapsedTime(&shared_milliseconds, shared_start, shared_stop);

    printf("Shared Memory Dot Product Result: %d\n", result[0]);
    printf("Shared Memory Time: %f milliseconds\n", shared_milliseconds);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_result);

    // Free allocated memory
    free(vectorA);
    free(vectorB);
    free(result);

    return 0;
}
